/* *********************************************************************
* DESCRIPTION :
* Serial Concurrent Wave Equation - C Version
* This program implements the concurrent wave equation
******************************************************************** */

#include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>
# include <math.h>
# include <time.h>
# define MAXPOINTS 1000000
# define MAXSTEPS 1000000
# define MINPOINTS 20
# define PI 3.14159265

# define blockSize 1024
void check_param ( void );
void init_line ( void );
void printfinal ( void );
int nsteps , /* number of time steps */
tpoints , /* total points along string */
rcode ; /* generic return code */
float values[MAXPOINTS+2], /* values at time t */
*values_d; 
/* *********************************************************************
* Checks input values from parameters
******************************************************************** */
void check_param ( void )
{
	char tchar [20];
	/* check number of points , number of iterations */
	while (( tpoints < MINPOINTS ) || ( tpoints > MAXPOINTS ) ) {
		printf ( " Enter number of points along vibrating string [% d -% d ]: ", MINPOINTS , MAXPOINTS );
		scanf ( " %s " , tchar ) ;
		tpoints = atoi ( tchar ) ;
		if (( tpoints < MINPOINTS ) || ( tpoints > MAXPOINTS ) )
			printf ( " Invalid . Please enter value between % d and % d \n " ,	MINPOINTS , MAXPOINTS );
	}
	while (( nsteps < 1) || ( nsteps > MAXSTEPS ) ) {
		printf ( " Enter number of time steps [1 -% d ]: " , MAXSTEPS );
		scanf ( " %s " , tchar ) ;
		nsteps = atoi ( tchar ) ;
		if (( nsteps < 1) || ( nsteps > MAXSTEPS ) )
			printf ( " Invalid . Please enter value between 1 and % d \n ", MAXSTEPS );
	}
	printf ( " Using points = %d , steps = %d \n " , tpoints , nsteps );
}
/* *********************************************************************
*initialization
******************************************************************** */
__global__ void init_line (int tpoints, float *values_d, float *oldval)
{    
	int i = 1 + blockIdx.x * blockSize + threadIdx.x;
	float x = (float) (i - 1) / (tpoints - 1);
	values_d[i] = __sinf(2.0 * PI * x);
	oldval[i] = values_d[i];
}
/* *********************************************************************
* merge three function
******************************************************************** */
__global__ void merge (int __tpoints, int __nsteps, float *__values_d)
{
	float oldval, newval, value;
    float dtime, c, dx, tau, sqtau;
	int i = 1 + blockIdx.x * blockSize + threadIdx.x;
	dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    float x = (float) (i - 1) / (__tpoints - 1);
	value = __sinf(2.0 * PI * x);
	oldval = value;
	
	if (i <= __tpoints) {
		for (int j = 1; j <= __nsteps; j++) {
			if ((i == 1) || (i == __tpoints ))
				newval = 0;
			else 
				newval = (2.0 * value) - oldval + (sqtau * -2.0 * value);
			oldval = value;
			value = newval;			
		}
		__values_d[i] = value;
	}	
}
/* *********************************************************************
* Print final results
******************************************************************** */
void printfinal ()
{
	int i ;
	for ( i = 1; i <= tpoints ; i ++) {
		printf("%6.4f ", values[i]);
		if (i % 10 == 0)
			printf("\n");
	}
}
/* *********************************************************************
* Main program
******************************************************************** */
int main ( int argc , char * argv [])
{
	sscanf ( argv [1] , "%d" ,& tpoints );
	sscanf ( argv [2] , "%d" ,& nsteps );
	
    int numBlocks = tpoints / blockSize +  !(tpoints % blockSize == 0);
    hipMalloc((void**) &values_d, sizeof(float) * (tpoints + 1));

	check_param ();
	printf("Initializing points on the line...\n");
	printf("Updating all points for all time steps...\n");
	
	merge<<<numBlocks, blockSize>>>(tpoints, nsteps, values_d);
	hipMemcpy(values, values_d, sizeof(float) * (tpoints + 1), hipMemcpyDeviceToHost);
	
	printf("Printing final results...\n");
	printfinal () ;
	printf("\nDone.\n\n");
	return 0;
}